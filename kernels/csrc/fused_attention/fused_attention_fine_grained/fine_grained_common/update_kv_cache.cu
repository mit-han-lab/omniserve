#include "hip/hip_runtime.h"
// Inspired by TRT-LLM.
// Modified by Haotian Tang and Shang Yang.
// @article{lin2024qserve,
//   title={QServe: W4A8KV4 Quantization and System Co-design for Efficient LLM Serving},
//   author={Lin*, Yujun and Tang*, Haotian and Yang*, Shang and Zhang, Zhekai and Xiao, Guangxuan and Gan, Chuang and Han, Song},
//   journal={arXiv preprint arXiv:2405.04532},
//   year={2024}
// }
// @article{yang2025lserve,
//   title={LServe: Efficient Long-sequence LLM Serving with Unified Sparse Attention},
//   author={Yang*, Shang and Guo*, Junxian and Tang, Haotian and Hu, Qinghao and Xiao, Guangxuan and Tang, Jiaming and Lin, Yujun and Liu, Zhijian and Lu, Yao and Han, Song},
//   year={2025}
// }
#include <torch/extension.h>
#include "ATen/cuda/HIPContext.h"
#include <c10/cuda/CUDAGuard.h>

#include "applyBiasRopeUpdateKVCache.h"

INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(half, KVBlockArray<false>, KVBlockArray<true>, true);
INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(half, KVBlockArray<false>, KVBlockArray<true>, false);
// INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(half, KVBlockArray, KVBlockArray, true);
// INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(half, KVBlockArray, KVBlockArray, false);
// INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(half, KVLinearBuffer, KVLinearBuffer, true);
// INSTANTIATE_ADDFUSEDQKVBIAS_TRANSPOSE(half, KVLinearBuffer, KVLinearBuffer, false);

void apply_bias_rope_update_kv_cache(const torch::Tensor qkv,
                                     torch::Tensor retrieval_seq_lens,
                                     c10::optional<torch::Tensor> streaming_seq_lens,
                                     torch::Tensor padding_offset,
                                    //  c10::optional<torch::Tensor> kv_pointers, // B x 2 x M
                                     c10::optional<torch::Tensor> retrieval_kv_pointers, // B x 2 x M
                                     c10::optional<torch::Tensor> streaming_kv_pointers, // B x 2 x M
                                     torch::Tensor retrieval_head_flags, // H
                                     torch::Tensor head_rank_table, // H
                                     // virtual sequence length (after padding)
                                     const int head_num,
                                     const int kv_head_num,
                                     const int seq_len,          // max seq len
                                     const int tokens_per_block, // default=64
                                    //  const int size_per_token,   // default = hidden_size * sizeof(dtype)
                                     const int size_per_retrieval_token,   // default = hidden_size * sizeof(dtype)
                                     const int size_per_streaming_token,   // default = hidden_size * sizeof(dtype)
                                     const int sink_token_num, const int local_token_num,
                                     const int sink_block_num, const int local_block_num,
                                     const int num_retrieval_kv_heads,
                                     const int num_streaming_kv_heads,
                                     const int rotary_embedding_dim,
                                     const float rotary_embedding_base,
                                     const float rotary_embedding_scale,
                                     const int rotary_embedding_max_positions,
                                     // neox_rotary_style = not interleaved
                                     const bool neox_rotary_style,
                                     const bool int4_kv_cache,
                                     const bool kv_cache_with_zeros
                                     )
{
    half *q_ptr = nullptr;
    half *qkv_ptr = reinterpret_cast<half *>(qkv.data_ptr<at::Half>());
    int *retrieval_seq_lens_ptr = retrieval_seq_lens.data_ptr<int>();
    int *retrieval_kv_seq_lens_ptr = retrieval_seq_lens_ptr;
    // TBD
    int *padding_offset_ptr = padding_offset.data_ptr<int>();
    half *qkv_bias_ptr = nullptr;
    int batch_size = retrieval_seq_lens.size(0);
    int retrieval_max_blocks_per_seq = retrieval_kv_pointers.has_value() ? retrieval_kv_pointers.value().size(-1) : 0;
    int streaming_max_blocks_per_seq = streaming_kv_pointers.has_value() ? streaming_kv_pointers.value().size(-1) : 0;
    KVBlockArray<false> retrievalkvTable(batch_size, retrieval_max_blocks_per_seq, tokens_per_block, size_per_retrieval_token, 0, 0, 0, 0, 0, 0);       // NOTE (Shang): The last two args are for hierarchical paging. Not sure if the current impl is correct. 
    KVBlockArray<true> streamingkvTable(batch_size, streaming_max_blocks_per_seq, tokens_per_block, size_per_streaming_token, sink_token_num, local_token_num, sink_block_num, local_block_num, 0, 0);
    
    retrievalkvTable.data = retrieval_kv_pointers.has_value() ? retrieval_kv_pointers.value().data_ptr<int64_t>() : nullptr;
    streamingkvTable.data = streaming_kv_pointers.has_value() ? streaming_kv_pointers.value().data_ptr<int64_t>() : nullptr;

    int *retrieval_head_flags_ptr = retrieval_head_flags.data_ptr<int>();
    int *head_rank_table_ptr = head_rank_table.data_ptr<int>();

    // NOTE: cyclic_kv_cache_len should not be 0.
    int cyclic_kv_cache_len = rotary_embedding_max_positions;
    int sink_token_len = 0;
    int token_num = qkv.size(0);
    // fix this
    int size_per_head = rotary_embedding_dim;
    RotaryScalingType rotary_scale_type = RotaryScalingType::kLINEAR;
    PositionEmbeddingType position_embedding_type = PositionEmbeddingType::kROPE_GPT_NEOX;
    int *medusa_position_offsets_ptr = nullptr;
    bool position_shift_enabled = false;
    float *scale_ptr = nullptr;
    int int8_mode = 1;
    KvCacheDataType cache_type;
    if (int4_kv_cache) {
        if (kv_cache_with_zeros)
        {
            cache_type = KvCacheDataType::ZINT4;
        }
        else
        {
            cache_type = KvCacheDataType::INT4;
        }
    }
    else {
        if (kv_cache_with_zeros)
        {
            cache_type = KvCacheDataType::ZINT8;
        }
        else
        {
            cache_type = KvCacheDataType::INT8;
        }
    }
    int beam_width = 1;
    bool enable_paged_kv_fmha = true;
    // TODO: grid_block_cache for different devices??
    int2 grid_block_cache = make_int2(96, 1024);
    auto stream = at::cuda::getCurrentCUDAStream();
    invokeApplyBiasRopeUpdateKVCache<half, KVBlockArray<false>, KVBlockArray<true>, false>(
        qkv_ptr, q_ptr, retrievalkvTable, streamingkvTable, retrieval_head_flags_ptr, head_rank_table_ptr, 
        qkv_bias_ptr, retrieval_seq_lens_ptr, retrieval_kv_seq_lens_ptr,
        padding_offset_ptr, batch_size, seq_len, cyclic_kv_cache_len,
        sink_token_len, token_num, head_num, kv_head_num, num_retrieval_kv_heads, num_streaming_kv_heads,
        size_per_head, rotary_embedding_dim, rotary_embedding_base,
        rotary_scale_type, rotary_embedding_scale, rotary_embedding_max_positions,
        position_embedding_type, medusa_position_offsets_ptr, position_shift_enabled,
        scale_ptr, int8_mode, cache_type,
        enable_paged_kv_fmha, beam_width, grid_block_cache, stream);

    // void invokeApplyBiasRopeUpdateKVCache<T, KVCacheBuffer, IS_GENERATE>(T * QKV, T * Q,
    //     KVCacheBuffer & kvTable, const T* qkv_bias, const int* seq_lens, const int* kv_seq_lens,
    //     const int* padding_offset, const int batch_size, const int seq_len, const int cyclic_kv_cache_len,
    //     const int sink_token_len, const int token_num, const int head_num, const int kv_head_num,
    //     const int size_per_head, const int rotary_embedding_dim, const float rotary_embedding_base,
    //     const RotaryScalingType rotary_scale_type, const float rotary_embedding_scale,
    //     const int rotary_embedding_max_positions, const PositionEmbeddingType position_embedding_type,
    //     const int* medusa_position_offsets, const bool position_shift_enabled, const float* scale,
    //     const int int8_mode, const KvCacheDataType cache_type, const float* kvScaleOrigQuant,
    //     const bool enable_paged_kv_fmha, const int beam_width, int2& grid_block_cache, hipStream_t stream);
}